#include "hip/hip_runtime.h"
#include "dens.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __matmul(
	float* a,
	float* b,
	float* c,
	const uint m,
	const uint n,
	const uint k
) {
	uint cx = blockIdx.x * blockDim.x + threadIdx.x;
	uint cy = blockIdx.y * blockDim.y + threadIdx.y;

	__shared__ float sm_a[BLOCK_SIZE_32 * BLOCK_SIZE_32];
	__shared__ float sm_b[BLOCK_SIZE_32 * BLOCK_SIZE_32];

	uint tidx = threadIdx.y * BLOCK_SIZE_32 + threadIdx.x;
	float val = 0.f;

	for (int i = 0; i < n; i += BLOCK_SIZE_32) {
		__syncthreads();

		sm_a[tidx] = (threadIdx.x + i) < n && cy < m ? a[cy * n + (threadIdx.x + i)] : 0.f;
		sm_b[tidx] = cx < k && (threadIdx.y + i) < n ? b[(threadIdx.y + i) * k + cx] : 0.f;

		__syncthreads();

		for (int e = 0; e < BLOCK_SIZE_32; ++e) {
			val += sm_a[threadIdx.y * BLOCK_SIZE_32 + e] * sm_b[e * BLOCK_SIZE_32 + threadIdx.x];
		}
	}

	if (cx < k && cy < m) {
		c[cy * k + cx] = val;
	}
}



/**********************************************/
/*											  */
/*				  host function 			  */
/*										      */
/**********************************************/

void check_dens(
	const Tensor& input,
	const Tensor& weight,
	const Tensor& output
) {
	if (input.n != output.n || input.c != weight.c || output.c != weight.n) {
		ErrorExcept(
			"[matmul_check] invalid matrix size input: %s, weight: %s, output: %s",
			dim_to_str(input),
			dim_to_str(weight),
			dim_to_str(output)
		);
	}
}

void dens(
	const hipStream_t st,
	const Tensor& input,
	const Tensor& weight,
	Tensor& output
) {
	check_dens(input, weight, output);

	dim3 threads(BLOCK_SIZE_32, BLOCK_SIZE_32);
	dim3 blocks = get_grid_size(threads, output.c, output.n);

	__matmul<<<blocks, threads, 0, st>>>(
		input.data,
		weight.data,
		output.data,
		input.n,
		weight.h,
		output.c
	);

	check_cuda(hipStreamSynchronize(st));
}
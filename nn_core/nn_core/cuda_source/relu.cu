#include "hip/hip_runtime.h"
#include "relu.cuh"

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <hip/device_functions.h>
#include <>


/**********************************************/
/*											  */
/*				 kernel function			  */
/*										      */
/**********************************************/

__global__ void __relu(
	float* a,
	float* b,
	const uint length
) {
	uint cx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (cx < length) {
		b[cx] = __max(0.f, a[cx]);
	}
}



/**********************************************/
/*											  */
/*				  host function 			  */
/*										      */
/**********************************************/

void relu(
	const Stream* stream,
	const Tensor* input,
	Tensor* output
) {
	size_t input_size = GetTotalSize(input);
	size_t output_size = GetTotalSize(output);

	if (input_size != output_size) {
		ErrorExcept("[relu] input�� output ����� �ȸ½��ϴ�. %d != %d", input_size, output_size);
	}

	int length = input->h * input->w * input->c;
	dim3 threads(BLOCK_SIZE);
	dim3 blocks(GetBlockSize(length));

	for (int i = 0; i < stream->st_size; ++i) {
		float* d_in = input->data + (i * length);
		float* d_out = output->data + (i * length);

		__relu<<<blocks, threads, 0, stream->st[i]>>>(
			d_in,
			d_out,
			length
		);
	}

	SyncStreams(stream);
}